#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

//NVIDIA GTX 670 1024 threads/block, 2.1B blocks	

hipError_t isPrimeWithCuda(long long int *a, long int *primes, int sizearray, int numprimes);
hipError_t whichPrimeWithCuda(long long int *a, long int *primes, int sizearray, int numprimes);
int findarraySize(int fibmaxval);
long long int* filltestarray(long int largeprime);
long int* addtolist(long int* oldlist, long long int newprime, int plength);

__global__ void isPrimeKernel(long long int *a, long int *primes, int sizearray, int numprimes)
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	long int idx = i*1024 + j;

	if (idx < sizearray)
	{
		a[idx] = a[0] + idx;
		for (int primeindx = 0; primeindx < numprimes; primeindx++)
		{
			if (a[idx] % primes[primeindx] == 0)
			{
				a[idx] = 0;
				return;
			}
		}
	}
}

__global__ void whichPrimeKernel(long long int *a, long int *primes, int sizearray, int numprimes)
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	long int idx = i * 1024 + j;

	if (idx < numprimes)
	{
		//a[idx] = a[0];
		//printf("Kernel: %d Checking: %lld %d", idx, a[0], primes[idx]);
		if (a[0] % primes[idx] == 0)
		{
			printf("Kernel: %d Prime Factor: %d\n",idx, primes[idx]);
			primes[idx] *= -1;
			return;
		}
	}
}

int main()
{
	int arraySize;
	double inputnum = 600851475143;
	long long int inputint = 600851475143;

	//arraySize = findarraySize(fibmaxval);
	double squareroot = std::sqrt(inputnum);
	arraySize = floor(squareroot);
	printf("Sqrt %d %f\n", arraySize, squareroot); fflush(stdout);

	long int *primelist;
	primelist = (long int *)malloc(sizeof(long int));
	primelist[0] = 2;

	int plength = 1;

	long long int *testarray;

	long int maxprime = primelist[plength-1];
	while (maxprime < arraySize)
	{
		int arraylength = maxprime * (maxprime - 1);
		printf("Arraylength: %d maxprime %d %d\n", arraylength, maxprime, primelist[plength-1]); fflush(stdout);
		testarray = filltestarray(maxprime);

		printf("Array: %d\n", testarray[0]); fflush(stdout);

		hipError_t cudaStatus = isPrimeWithCuda(testarray, primelist, arraylength, plength);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}

		for (int i = 0; i < arraylength; i++)
		{
			if (testarray[i] != 0)
			{
				primelist = addtolist(primelist, testarray[i], plength);
				plength += 1;
				//printf("Add %d to total primes %d\n", primelist[plength-1], plength); fflush(stdout);
			}
		}

		free(testarray);
		maxprime = primelist[plength-1];
		printf("Max prime %d %d\n", primelist[plength-1], plength); fflush(stdout);
	}

	testarray = (long long int *)malloc(sizeof(long long int));
	int arraylength = 1;
	printf("Final Run arraylength %d %lld\n", plength, inputint); fflush(stdout);
	testarray[0] = inputint;
	printf("Testarray %lld\n", testarray[0]); fflush(stdout);
	hipError_t cudaStatus = whichPrimeWithCuda(testarray, primelist, arraylength, plength);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	long int biggest = 0;
	for (int i = 0; i < plength; i++)
	{
		if (primelist[i] < 0)
		{
			biggest = primelist[i];
		}
	}
	printf("Biggest prime %d", biggest);
	free(testarray);
	//printf("testarray %d %d\n",testarray[0],testarray[1]);
	//hipError_t cudaStatus = findMultipleWithCuda(a, arraySize);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "addWithCuda failed!");
	//	return 1;
	//}

	//printf("Post {%d,%d,%d,%d,%d}\n",
	//	a[0], a[1], a[2], a[3], a[4]); fflush(stdout);

	//int result = 0;
	////	for (auto& num : a)
	//for (int i = 0; i < arraySize; i++)
	//{
	//	if (a[i] != 0)
	//	{
	//		result = a[i];
	//	}
	//}

	//printf("Result %d", result);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	//cudaStatus = hipDeviceReset();
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipDeviceReset failed!");
	//	return 1;
	//}

	return 0;
}

long long int* filltestarray(long int largeprime)
{
	long long int *array;
	long int maxval = largeprime * largeprime - largeprime;

	printf("Filltestarray maxval %d largeprime %d\n", maxval, largeprime);
	array = (long long  int *)malloc(sizeof(long long int)*maxval);
	array[0] = largeprime+1;
	return array;
}

long int* addtolist(long int* oldlist, long long int newprime, int plength)
{
	oldlist = (long int *)realloc(oldlist, sizeof(long int)*(plength+1));
	oldlist[plength] = newprime;
	return oldlist;
}

int findarraySize(int fibmaxval)
{
	int indx = 2;
	int prev1 = 2;
	int prev2 = 1;
	int currentval = 0;

	while (currentval < fibmaxval)
	{
		indx += 1;
		currentval = prev1 + prev2;
		prev2 = prev1;
		prev1 = currentval;
		printf("%d,%d\n", indx, currentval);
	}
	return indx - 1;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t isPrimeWithCuda(long long int *a, long int *primes, int sizearray, int numprimes)
{
	long long int *dev_a = 0;
	long int *dev_primes = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_a, sizearray * sizeof(long long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_primes, numprimes * sizeof(long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, sizeof(long long int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_primes, primes, numprimes*sizeof(long int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	int numblocks = (sizearray / 1024)+1;
	printf("Cuda blocks %d\n", numblocks); fflush(stdout);
	printf("Sizearray %d numprimes %d\n", sizearray, numprimes); fflush(stdout);

	// Launch a kernel on the GPU with one thread for each element.
	isPrimeKernel << <numblocks, 1024 >> >(dev_a, dev_primes, sizearray, numprimes);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(a, dev_a, sizearray * sizeof(long long int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_primes);

	return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t whichPrimeWithCuda(long long int *a, long int *primes, int sizearray, int numprimes)
{
	long long int *dev_a = 0;
	long int *dev_primes = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_a, sizearray * sizeof(long long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_primes, numprimes * sizeof(long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, sizeof(long long int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_primes, primes, numprimes*sizeof(long int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	int numblocks = (numprimes / 1024) + 1;
	printf("Cuda blocks %d\n", numblocks); fflush(stdout);
	printf("Sizearray %d numprimes %d\n", sizearray, numprimes); fflush(stdout);

	// Launch a kernel on the GPU with one thread for each element.
	whichPrimeKernel << <numblocks, 1024 >> >(dev_a, dev_primes, sizearray, numprimes);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(a, dev_a, sizearray * sizeof(long long int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(primes, dev_primes, numprimes* sizeof(long int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_primes);

	return cudaStatus;
}
